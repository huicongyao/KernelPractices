#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <vector>

#include "../utils.hpp"

void cublas_sgemm(float *A, float *B, float *C, size_t M, size_t N, size_t K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);
  // hipDeviceSynchronize();
}

int main() {
  constexpr int M = 5120, N = 5120, K = 5120;
  constexpr int repeat = 10;

  printf("Running GEMM benchmarks with M=%d, N=%d, K=%d\n", M, N, K);

  // Benchmark naive sgemm
  benchmark_gemm(cublas_sgemm, M, N, K, "cublas_sgemm", repeat);
}