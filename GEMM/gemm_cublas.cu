#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <mma.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <vector>

#include "../utils.hpp"

void cublas_sgemm(float *A, float *B, float *C, size_t M, size_t N, size_t K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               HIPBLAS_GEMM_DEFAULT);
  // hipDeviceSynchronize();
}

void cublas_sgemm_tf32(float *A, float *B, float *C, size_t M, size_t N,
                       size_t K) {
  hipblasHandle_t handle = nullptr;
  hipblasCreate(&handle);
  hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

  static float alpha = 1.0;
  static float beta = 0.0;

  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B, HIP_R_32F,
               N, A, HIP_R_32F, K, &beta, C, HIP_R_32F, N, HIPBLAS_COMPUTE_32F,
               CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}

void benchmark_group_gemm(int M, int N, int K, int repeats = 10) {
  printf("Running GEMM benchmarks with M=%d, N=%d, K=%d\n", M, N, K);
  // benchmark_gemm(cublas_sgemm, M, N, K, "cublas_sgemm warp up run",
  //                repeats / 5);
  // benchmark_gemm(cublas_sgemm, M, N, K, "cublas_sgemm", repeats);

  benchmark_gemm(cublas_sgemm_tf32, M, N, K, "cublas_sgemm_tf32", repeats);
}

int main() {
  constexpr int repeats = 30;
  std::vector<int> shape = {4096, 8192};
  // std::vector<int> shape = {5120};
  for (auto M : shape) {
    for (auto N : shape) {
      for (auto K : shape) {
        benchmark_group_gemm(M, N, K, repeats);
      }
    }
  }

  return 0;
}